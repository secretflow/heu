#include "hip/hip_runtime.h"
// Copyright 2023 Ant Group Co., Ltd.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "heu/library/algorithms/paillier_gpu/gpulib/error.h"
#include "heu/library/algorithms/paillier_gpu/gpulib/gpu_paillier.h"
#include "heu/library/algorithms/paillier_gpu/gpulib/gpupaillier.h"

template <class params>
__device__ __forceinline__ void paillier_t<params>::fixed_window_powm_odd(
    bn_t &result, const bn_t &x, const bn_t &power, const bn_t &modulus) {
  bn_t t;
  bn_local_t window[1 << window_bits];
  int32_t index, position, offset;
  uint32_t np0;

  // conmpute x^power mod modulus, using the fixed window algorithm
  // requires:  x<modulus,  modulus is odd
  // compute x^0 (in Montgomery space, this is just 2^BITS - modulus)
  cgbn_negate(_env, t, modulus);
  cgbn_store(_env, window + 0, t);

  // convert x into Montgomery space, store into window table
  np0 = cgbn_bn2mont(_env, result, x, modulus);
  cgbn_store(_env, window + 1, result);
  cgbn_set(_env, t, result);

// compute x^2, x^3, ... x^(2^window_bits-1), store into window table
#pragma nounroll
  for (index = 2; index < (1 << window_bits); index++) {
    cgbn_mont_mul(_env, result, result, t, modulus, np0);
    cgbn_store(_env, window + index, result);
  }

  // find leading high bit
  position = params::BITS - cgbn_clz(_env, power);

  // break the exponent into chunks, each window_bits in length
  // load the most significant non-zero exponent chunk
  offset = position % window_bits;
  if (offset == 0)
    position = position - window_bits;
  else
    position = position - offset;
  index = cgbn_extract_bits_ui32(_env, power, position, window_bits);
  cgbn_load(_env, result, window + index);

  // process the remaining exponent chunks
  while (position > 0) {
// square the result window_bits times
#pragma nounroll
    for (int sqr_count = 0; sqr_count < window_bits; sqr_count++)
      cgbn_mont_sqr(_env, result, result, modulus, np0);

    // multiply by next exponent chunk
    position = position - window_bits;
    index = cgbn_extract_bits_ui32(_env, power, position, window_bits);
    cgbn_load(_env, t, window + index);
    cgbn_mont_mul(_env, result, result, t, modulus, np0);
  }

  // we've processed the exponent now, convert back to normal space
  cgbn_mont2bn(_env, result, result, modulus, np0);
}

// Sum the encrypted values by multiplying the ciphertexts
template <class params>
__global__ void kernel_paillier_enc(cgbn_error_report_t *report,
                                    gpu_paillier_ciphertext_t *gpu_res,
                                    gpu_paillier_pubkey_t *gpu_pub,
                                    gpu_paillier_plaintext_t *gpu_pt,
                                    gpu_paillier_random_t *rand,
                                    uint32_t count) {
  // decode an instance number from the blockIdx and threadIdx
  int32_t i;
  i = (blockIdx.x * blockDim.x + threadIdx.x) / params::TPI;
  if (i >= count) return;

  // 4096bit variables
  paillier_t<params> po(cgbn_report_monitor, report, i);
  typename paillier_t<params>::bn_t r1, r2, g, x, p, m, n;
  typename paillier_t<params>::bn_wide_t r;

  cgbn_load(po._env, g, &(gpu_pub->n_plusone));
  cgbn_load(po._env, m, &(gpu_pt[i].m));
  cgbn_load(po._env, n, &(gpu_pub->n_squared));

  // Fake code:paillier.fixed_window_powm_odd(gpu_res[i].c, gpu_pub->n_plusone,
  // gpu_pt[i].m, gpu_pub->n_squared);
  po.fixed_window_powm_odd(r1, g, m, n);

  cgbn_load(po._env, x, &(rand[i].m));
  cgbn_load(po._env, p, &(gpu_pub->n));

  // Fake code:paillier.fixed_window_powm_odd(gpu_x.x, rand[i].m, gpu_pub->n,
  // gpu_pub->n_squared);
  po.fixed_window_powm_odd(r2, x, p, n);
  // cgbn_modular_power(po._env,r2,x,p,n); //the x should less than n

  // Fake code:paillier.mul(&(gpu_res[i].c), &(gpu_res[i].c), &(gpu_x.x));
  cgbn_mul_wide(po._env, r, r1, r2);  // the r is 8192 bit,r=r1*r2;

  // Fake code:paillier.mod(&(gpu_res[i].c), &(gpu_pub->n_squared));
  cgbn_rem_wide(po._env, r1, r,
                n);  // back to 4096 bit for next mod ,r1=r%m ,the high CGBN of
                     // r is less than the denominator

  cgbn_store(po._env, &(gpu_res[i].c), r1);
  return;
}

template <class params>
__global__ void kernel_paillier_dec(cgbn_error_report_t *report,
                                    gpu_paillier_plaintext_t *gpu_res,
                                    gpu_paillier_pubkey_t *gpu_pub,
                                    gpu_paillier_prvkey_t *gpu_prv,
                                    gpu_paillier_ciphertext_t *gpu_ct,
                                    uint32_t count) {
  int32_t i;
  i = (blockIdx.x * blockDim.x + threadIdx.x) / params::TPI;
  if (i >= count) return;

  paillier_t<params> po(cgbn_report_monitor, report, i);
  typename paillier_t<params>::bn_t r, c, l, n, p, x;
  typename paillier_t<params>::bn_wide_t dr;

  cgbn_load(po._env, c, &(gpu_ct[i].c));
  cgbn_load(po._env, l, &(gpu_prv->lambda));
  cgbn_load(po._env, x, &(gpu_prv->x));
  cgbn_load(po._env, n, &(gpu_pub->n_squared));
  cgbn_load(po._env, p, &(gpu_pub->n));

  // Fake code:paillier.fixed_window_powm_odd(gpu_res[i].m, gpu_ct[i].c,
  // gpu_prv[i].lambda, gpu_pub[i].n_squared);
  po.fixed_window_powm_odd(r, c, l, n);
  // Fake code:paillier._env.sub_ui32(gpu_res[i].m, gpu_res[i].m, 1);
  po._env.sub_ui32(r, r, 1);
  // Fake code:paillier.div(gpu_res[i].m, gpu_res[i].m, gpu_pub->n);
  cgbn_div(po._env, r, r, p);
  // Fake code:paillier.mul(gpu_res[i].m, gpu_res[i].m, gpu_prv->x);
  cgbn_mul_wide(po._env, dr, r, x);  // 8192bits,should be fixed
  // Fake code:paillier.mod(gpu_res[i].m, gpu_pub->n);
  cgbn_rem_wide(po._env, r, dr, p);  // back to 4096,the high CGBN of num is
                                     // less than the denominator, denom.
  cgbn_store(po._env, &(gpu_res[i].m), r);
  return;
}

// Sum the encrypted values by multiplying the ciphertexts
template <class params>
__global__ void kernel_paillier_e_add(cgbn_error_report_t *report,
                                      gpu_paillier_ciphertext_t *gpu_res,
                                      gpu_paillier_pubkey_t *gpu_pub,
                                      gpu_paillier_ciphertext_t *gpu_ct0,
                                      gpu_paillier_ciphertext_t *gpu_ct1,
                                      uint32_t count) {
  int32_t i;
  i = (blockIdx.x * blockDim.x + threadIdx.x) / params::TPI;
  if (i >= count) return;

  paillier_t<params> po(cgbn_report_monitor, report, i);

  typename paillier_t<params>::bn_t r, c0, c1, n;
  typename paillier_t<params>::bn_wide_t dr;
  cgbn_load(po._env, c0, &(gpu_ct0[i].c));
  cgbn_load(po._env, c1, &(gpu_ct1[i].c));
  cgbn_load(po._env, n, &(gpu_pub->n_squared));
  // paillier.d_mul(gpu_res[i].c, gpu_ct0[i].c, gpu_ct1[i].c);
  cgbn_mul_wide(po._env, dr, c0, c1);  // dr=c0*c1,  dr is 8192bits
  // paillier.d_mod(gpu_res[i].c, gpu_pub->n_squared);
  cgbn_rem_wide(po._env, r, dr, n);  // back to 4096
  cgbn_store(po._env, &(gpu_res[i].c), r);
  return;
}

template <class params>
__global__ void kernel_paillier_e_sub(cgbn_error_report_t *report,
                                      gpu_paillier_ciphertext_t *gpu_res,
                                      gpu_paillier_pubkey_t *gpu_pub,
                                      gpu_paillier_ciphertext_t *gpu_ct0,
                                      gpu_paillier_ciphertext_t *gpu_ct1,
                                      uint32_t count) {
  int32_t i;
  i = (blockIdx.x * blockDim.x + threadIdx.x) / params::TPI;
  if (i >= count) return;

  paillier_t<params> po(cgbn_report_monitor, report, i);

  typename paillier_t<params>::bn_t r, c0, c1, n;
  typename paillier_t<params>::bn_wide_t dr;
  cgbn_load(po._env, c0, &(gpu_ct0[i].c));
  cgbn_load(po._env, c1, &(gpu_ct1[i].c));
  cgbn_load(po._env, n, &(gpu_pub->n_squared));

  cgbn_modular_inverse(po._env, r, c1, n);  // r=inv(c1)

  cgbn_mul_wide(po._env, dr, c0, r);  // dr=c0*r,  dr is 8192bits
  cgbn_rem_wide(po._env, r, dr, n);   // back to 4096
  cgbn_store(po._env, &(gpu_res[i].c), r);
  return;
}

template <class params>
__global__ void kernel_paillier_e_sub_ctpt(cgbn_error_report_t *report,
                                           gpu_paillier_ciphertext_t *gpu_res,
                                           gpu_paillier_pubkey_t *gpu_pub,
                                           gpu_paillier_ciphertext_t *gpu_ct,
                                           gpu_paillier_plaintext_t *gpu_pt,
                                           uint32_t count) {
  int32_t i;
  i = (blockIdx.x * blockDim.x + threadIdx.x) / params::TPI;
  if (i >= count) return;

  paillier_t<params> po(cgbn_report_monitor, report, i);
  typename paillier_t<params>::bn_t r, c, m, n, g, ri, ro;
  typename paillier_t<params>::bn_wide_t dr;
  cgbn_load(po._env, c, &(gpu_ct[i].c));
  cgbn_load(po._env, m, &(gpu_pt[i].m));
  cgbn_load(po._env, n, &(gpu_pub->n_squared));
  cgbn_load(po._env, g, &(gpu_pub->n_plusone));

  po.fixed_window_powm_odd(r, g, m, n);

  cgbn_modular_inverse(po._env, ri, r, n);  // ri=inv(r)

  cgbn_mul_wide(po._env, dr, c, ri);  // dr=c*ri,  dr is 8192bits

  cgbn_rem_wide(po._env, r, dr, n);  // back to 4096
  cgbn_store(po._env, &(gpu_res[i].c), r);

  return;
}

template <class params>
__global__ void kernel_paillier_e_sub_ptct(cgbn_error_report_t *report,
                                           gpu_paillier_ciphertext_t *gpu_res,
                                           gpu_paillier_pubkey_t *gpu_pub,
                                           gpu_paillier_plaintext_t *gpu_pt,
                                           gpu_paillier_ciphertext_t *gpu_ct,
                                           uint32_t count) {
  int32_t i;
  i = (blockIdx.x * blockDim.x + threadIdx.x) / params::TPI;
  if (i >= count) return;

  paillier_t<params> po(cgbn_report_monitor, report, i);
  typename paillier_t<params>::bn_t r, c, m, n, g, ri, rm;
  typename paillier_t<params>::bn_wide_t dr;
  cgbn_load(po._env, c, &(gpu_ct[i].c));
  cgbn_load(po._env, m, &(gpu_pt[i].m));
  cgbn_load(po._env, n, &(gpu_pub->n_squared));
  cgbn_load(po._env, g, &(gpu_pub->n_plusone));

  po.fixed_window_powm_odd(r, g, m, n);  // r=g^m mod n;

  cgbn_modular_inverse(po._env, ri, c, n);  // ri=inv(c)

  cgbn_mul_wide(po._env, dr, r, ri);  // dr=r*ri,  dr is 8192bits
  cgbn_rem_wide(po._env, r, dr, n);   // back to 4096
  cgbn_store(po._env, &(gpu_res[i].c), r);
  return;
}

// inv
template <class params>
__global__ void kernel_paillier_inv(cgbn_error_report_t *report,
                                    gpu_paillier_ciphertext_t *gpu_res,
                                    gpu_paillier_pubkey_t *gpu_pub,
                                    gpu_paillier_ciphertext_t *gpu_ctx,
                                    uint32_t count) {
  int32_t i;
  i = (blockIdx.x * blockDim.x + threadIdx.x) / params::TPI;
  if (i >= count) return;

  paillier_t<params> po(cgbn_report_monitor, report, i);
  typename paillier_t<params>::bn_t r, c, n;
  // typename paillier_t<params>::bn_wide_t  dr;
  cgbn_load(po._env, c, &(gpu_ctx[i].c));
  cgbn_load(po._env, n, &(gpu_pub->n_squared));

  cgbn_modular_inverse(po._env, r, c, n);

  cgbn_store(po._env, &(gpu_res[i].c), r);
  return;
}

// inv inplace , it can not work, because the memory is not managed by the GPU
template <class params>
__global__ void kernel_paillier_inv_inplace(cgbn_error_report_t *report,
                                            gpu_paillier_pubkey_t *gpu_pub,
                                            gpu_paillier_ciphertext_t *gpu_ctx,
                                            uint32_t count) {
  int32_t i;
  i = (blockIdx.x * blockDim.x + threadIdx.x) / params::TPI;
  if (i >= count) return;

  paillier_t<params> po(cgbn_report_monitor, report, i);
  typename paillier_t<params>::bn_t r, c, n;
  cgbn_load(po._env, c, &(gpu_ctx[i].c));
  cgbn_load(po._env, n, &(gpu_pub->n_squared));

  cgbn_modular_inverse(po._env, r, c, n);

  cgbn_store(po._env, &(gpu_ctx[i].c), r);  // replace the inpute
  return;
}

template <class params>
__global__ void kernel_paillier_e_add_const(cgbn_error_report_t *report,
                                            gpu_paillier_ciphertext_t *gpu_res,
                                            gpu_paillier_pubkey_t *gpu_pub,
                                            gpu_paillier_ciphertext_t *gpu_ct,
                                            gpu_paillier_plaintext_t *gpu_con,
                                            uint32_t count) {
  int32_t i;
  i = (blockIdx.x * blockDim.x + threadIdx.x) / params::TPI;
  if (i >= count) return;

  paillier_t<params> po(cgbn_report_monitor, report, i);
  typename paillier_t<params>::bn_t r, c, n, g, t;
  typename paillier_t<params>::bn_wide_t dr;

  cgbn_load(po._env, c, &(gpu_ct[i].c));
  cgbn_load(po._env, n, &(gpu_pub->n_squared));
  cgbn_load(po._env, g, &(gpu_pub->n_plusone));
  cgbn_load(po._env, t, &(gpu_con[i].m));

  // Fake code: po.d_fixed_window_powm_odd(gpu_res[i].c, gpu_pub->n_plusone,
  // gpu_con[i], gpu_pub->n_squared);
  po.fixed_window_powm_odd(r, g, t, n);
  // Fake code: po.d_mul(gpu_res[i].c, gpu_ct[i].c, gpu_res[i].c);
  cgbn_mul_wide(po._env, dr, c, r);  // dr=c0*c1,  dr is 8192bits
  // Fake code: po.d_mod(gpu_res[i].c,gpu_pub.n_squared);
  cgbn_rem_wide(po._env, r, dr, n);  // back to 4096
  cgbn_store(po._env, &(gpu_res[i].c), r);
  return;
}

template <class params>
__global__ void kernel_paillier_e_mul_const(cgbn_error_report_t *report,
                                            gpu_paillier_ciphertext_t *gpu_res,
                                            gpu_paillier_pubkey_t *gpu_pub,
                                            gpu_paillier_ciphertext_t *gpu_ct,
                                            gpu_paillier_plaintext_t *gpu_con,
                                            uint32_t count) {
  int32_t i;
  i = (blockIdx.x * blockDim.x + threadIdx.x) / params::TPI;
  if (i >= count) return;

  paillier_t<params> po(cgbn_report_monitor, report, i);
  typename paillier_t<params>::bn_t r, c, n, t;
  cgbn_load(po._env, c, &(gpu_ct[i].c));
  cgbn_load(po._env, n, &(gpu_pub->n_squared));
  cgbn_load(po._env, t, &(gpu_con[i].m));
  // Fake code:paillier.d_fixed_window_powm_odd(gpu_res[i].c, gpu_ct[i].c,
  // gpu_con[i].m, gpu_pub->n_squared);
  po.fixed_window_powm_odd(r, c, t, n);
  cgbn_store(po._env, &(gpu_res[i].c), r);
  return;
}

template <class params>
__global__ void kernel_paillier_compare(cgbn_error_report_t *report,
                                        gpu_paillier_plaintext_t *gpu_plain,
                                        uint32_t *gpu_res, uint32_t count) {
  int32_t i;
  i = (blockIdx.x * blockDim.x + threadIdx.x) / params::TPI;
  if (i >= count) return;

  int32_t j = -1;
  paillier_t<params> po(cgbn_report_monitor, report, i);
  typename paillier_t<params>::bn_t r;
  cgbn_load(po._env, r, &(gpu_plain[i].m));
  j = cgbn_compare_ui32(
      po._env, r, gpu_res[i]);  // compare the gpu result and the cpu result
  if (j != 0) printf("instance %d error: %u \n", i, gpu_res[i]);
  return;
  return;
}

void cudainit() {
  int count;
  hipGetDeviceCount(&count);
  hipError_t error_t = hipSetDevice(0);
  if (error_t != hipSuccess) printf("cuda error\n");
  error_t = hipDeviceSetCacheConfig(hipFuncCachePreferL1);
  if (error_t != hipSuccess) printf("cuda error\n");
}

//*********************************************gpu
// api*****************************************
int gpu_paillier_enc_bk(h_paillier_ciphertext_t *res, h_paillier_pubkey_t *pub,
                        h_paillier_plaintext_t *pt, h_paillier_random_t *rand,
                        unsigned int count) {
  int32_t TPB = (params::TPB == 0)
                    ? 64
                    : params::TPB;  // default threads per block to 128
  int32_t TPI = params::TPI, IPB = TPB / TPI;

  unsigned int ps, BPG;
  BPG = 256;
  ps = TPB * BPG;  // kernel parallel ,256 means blocks per Grid

  unsigned int rem, sm_num, sm_count, sm_count_tail, loop, i = 0, j = 0;
  sm_num = 3;  // 3 streams is enough
  if (count < 3) {
    sm_num = 1;
  }
  sm_count = count / sm_num;  // sm_count may be bigger than ps
  sm_count_tail =
      count -
      sm_count * sm_num;  // it is very important ,the count number should be
                          // the multiples of 3, if not it will left 1，2。
  // sm_count_malloc=sm_count+sm_count_tail;//+2 for the left data,it's depends
  loop = sm_count /
         ps;  // loop could be 0, loops for each stream,every time is ps.
  if (sm_num == 1) {
    loop = 0;
  }

  cudainit();
  hipStream_t stream[sm_num];
  // create stream
  for (i = 0; i < sm_num; i++) {
    hipStreamCreate(&(stream[i]));
  }

  // malloc for each stream
  gpu_paillier_ciphertext_t *gpu_result[sm_num];
  gpu_paillier_pubkey_t *gpu_pub[sm_num];
  gpu_paillier_plaintext_t *gpu_pt[sm_num];
  gpu_paillier_random_t *gpu_random[sm_num];
  cgbn_error_report_t *report[sm_num];
  for (i = 0; i < sm_num; i++) {
    CUDA_CHECK(hipMalloc((void **)&gpu_result[i],
                          sizeof(gpu_paillier_ciphertext_t) * ps));
    CUDA_CHECK(hipMalloc((void **)&gpu_pub[i], sizeof(gpu_paillier_pubkey_t)));
    CUDA_CHECK(
        hipMalloc((void **)&gpu_pt[i], sizeof(gpu_paillier_plaintext_t) * ps));
    CUDA_CHECK(hipMalloc((void **)&gpu_random[i],
                          sizeof(gpu_paillier_random_t) * ps));
    CUDA_CHECK(cgbn_error_report_alloc(&report[i]));
  }

  if (loop == 0) {
    rem = sm_count;
  } else {
    rem = sm_count - loop * ps;  // each stream has rem, except count is < 3.
  }

  printf("sm_count:%d, sm_count_tail:%d,loop:%d,sm_Num:%d,rem:%d,ps:%d\n",
         sm_count, sm_count_tail, loop, sm_num, rem, ps);
  for (i = 0; i < loop; i++)  // all it ps,the wave
  {
    for (j = 0; j < sm_num; j++) {
      hipMemcpyAsync((void *)(gpu_pt[j]),
                      (pt + sm_num * i * ps * sizeof(gpu_paillier_plaintext_t) +
                       j * ps * sizeof(gpu_paillier_plaintext_t)),
                      sizeof(gpu_paillier_plaintext_t) * ps,
                      hipMemcpyHostToDevice, stream[j]);
      hipMemcpyAsync((void *)gpu_pub[j], pub, sizeof(gpu_paillier_pubkey_t),
                      hipMemcpyHostToDevice, stream[j]);
      hipMemcpyAsync((void *)(gpu_random[j]),
                      (rand + sm_num * i * ps * sizeof(gpu_paillier_random_t) +
                       j * ps * sizeof(gpu_paillier_random_t)),
                      sizeof(gpu_paillier_random_t) * ps,
                      hipMemcpyHostToDevice, stream[j]);
      // kernel_paillier_enc<params>  <<< BPG, TPB,0,stream[j]  >>> (report[i],
      // gpu_result[j], gpu_pub[j], gpu_pt[j], gpu_random[j], ps);
      kernel_paillier_enc<params><<<(ps + IPB - 1) / IPB, TPB, 0, stream[j]>>>(
          report[i], gpu_result[j], gpu_pub[j], gpu_pt[j], gpu_random[j], ps);

      hipMemcpyAsync(
          (void *)(res + sm_num * i * ps * sizeof(gpu_paillier_ciphertext_t) +
                   j * ps * sizeof(gpu_paillier_ciphertext_t)),
          (gpu_result[j]), sizeof(gpu_paillier_ciphertext_t) * ps,
          hipMemcpyDeviceToHost, stream[j]);
      printf("loop!=0: %d,stream number:%d\n", i, j);
    }
  }

  i = loop;
  for (j = 0; j < sm_num; j++)  // the rem ,works well
  {
    hipMemcpyAsync((void *)(gpu_pt[j]),
                    (pt + sm_num * i * ps * sizeof(gpu_paillier_plaintext_t) +
                     j * rem * sizeof(gpu_paillier_plaintext_t)),
                    sizeof(gpu_paillier_plaintext_t) * rem,
                    hipMemcpyHostToDevice, stream[j]);
    hipMemcpyAsync((void *)gpu_pub[j], pub, sizeof(gpu_paillier_pubkey_t),
                    hipMemcpyHostToDevice, stream[j]);
    hipMemcpyAsync((void *)(gpu_random[j]),
                    (rand + sm_num * i * ps * sizeof(gpu_paillier_random_t) +
                     j * rem * sizeof(gpu_paillier_random_t)),
                    sizeof(gpu_paillier_random_t) * rem, hipMemcpyHostToDevice,
                    stream[j]);
    // kernel_paillier_enc<params>  <<< BPG, TPB,0,stream[j] >>> (report[i],
    // gpu_result[j], gpu_pub[j], gpu_pt[j], gpu_random[j], rem);
    kernel_paillier_enc<params><<<(rem + IPB - 1) / IPB, TPB, 0, stream[j]>>>(
        report[i], gpu_result[j], gpu_pub[j], gpu_pt[j], gpu_random[j], rem);

    hipMemcpyAsync(
        (void *)(res + sm_num * i * ps * sizeof(gpu_paillier_ciphertext_t) +
                 j * rem * sizeof(gpu_paillier_ciphertext_t)),
        (gpu_result[j]), sizeof(gpu_paillier_ciphertext_t) * rem,
        hipMemcpyDeviceToHost, stream[j]);
    printf("loop=i: %d,stream number:%d\n", i, j);
  }
  j = 0;
  if (sm_count_tail > 0) {
    hipMemcpyAsync((void *)(gpu_pt[j]),
                    (pt + sm_num * i * ps * sizeof(gpu_paillier_plaintext_t) +
                     sm_num * rem * sizeof(gpu_paillier_plaintext_t)),
                    sizeof(gpu_paillier_plaintext_t) * sm_count_tail,
                    hipMemcpyHostToDevice, stream[j]);
    hipMemcpyAsync((void *)gpu_pub[j], pub, sizeof(gpu_paillier_pubkey_t),
                    hipMemcpyHostToDevice, stream[j]);
    hipMemcpyAsync((void *)(gpu_random[j]),
                    (rand + sm_num * i * ps * sizeof(gpu_paillier_random_t) +
                     sm_num * rem * sizeof(gpu_paillier_random_t)),
                    sizeof(gpu_paillier_random_t) * sm_count_tail,
                    hipMemcpyHostToDevice, stream[j]);
    // kernel_paillier_enc<params> <<< BPG, TPB,0,stream[j] >>> (report[i],
    // gpu_result[j], gpu_pub[j], gpu_pt[j], gpu_random[j], sm_count_tail);
    kernel_paillier_enc<params>
        <<<(sm_count_tail + IPB - 1) / IPB, TPB, 0, stream[j]>>>(
            report[i], gpu_result[j], gpu_pub[j], gpu_pt[j], gpu_random[j],
            sm_count_tail);

    hipMemcpyAsync(
        (void *)(res + sm_num * i * ps * sizeof(gpu_paillier_ciphertext_t) +
                 sm_num * rem * sizeof(gpu_paillier_ciphertext_t)),
        (gpu_result[j]), sizeof(gpu_paillier_ciphertext_t) * sm_count_tail,
        hipMemcpyDeviceToHost, stream[j]);
    printf("sm_count_tail ,stream number:%d\n", j);
  }
  for (j = 0; j < sm_num; j++) {
    hipStreamSynchronize(stream[j]);
  }
  for (int j = 0; j < sm_num; j++) {
    hipStreamDestroy(stream[j]);
  }
  CUDA_LAST_CHECK();
  for (i = 0; i < sm_num; i++) {
    CUDA_CHECK(hipFree(gpu_pt[i]));      // data_in_gpu
    CUDA_CHECK(hipFree(gpu_pub[i]));     // pub
    CUDA_CHECK(hipFree(gpu_random[i]));  // random data
    CUDA_CHECK(hipFree(gpu_result[i]));  // cm
    CGBN_CHECK(report[i]);
    CUDA_CHECK(cgbn_error_report_free(report[i]));
  }

  return 0;
}

int gpu_paillier_enc(h_paillier_ciphertext_t *res, h_paillier_pubkey_t *pub,
                     h_paillier_plaintext_t *pt, h_paillier_random_t *rand,
                     unsigned int count) {
  int32_t TPB = (params::TPB == 0)
                    ? 64
                    : params::TPB;  // default threads per block to 128
  int32_t TPI = params::TPI, IPB = TPB / TPI;

  gpu_paillier_ciphertext_t *gpu_result;
  gpu_paillier_pubkey_t *gpu_pub;
  gpu_paillier_plaintext_t *gpu_pt;
  gpu_paillier_random_t *gpu_random;
  cgbn_error_report_t *report;

  int32_t BPG = 256;

  CUDA_CHECK(hipSetDevice(0));
  CUDA_CHECK(hipMalloc((void **)&gpu_result,
                        sizeof(gpu_paillier_ciphertext_t) * count));
  CUDA_CHECK(hipMalloc((void **)&gpu_pub, sizeof(gpu_paillier_pubkey_t)));
  CUDA_CHECK(
      hipMalloc((void **)&gpu_pt, sizeof(gpu_paillier_plaintext_t) * count));
  CUDA_CHECK(
      hipMalloc((void **)&gpu_random, sizeof(gpu_paillier_random_t) * count));
  CUDA_CHECK(cgbn_error_report_alloc(&report));

  CUDA_CHECK(hipMemcpy(gpu_pub, pub, sizeof(gpu_paillier_pubkey_t),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(gpu_pt, (gpu_paillier_plaintext_t *)pt,
                        sizeof(gpu_paillier_plaintext_t) * count,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(gpu_random, (gpu_paillier_random_t *)rand,
                        sizeof(gpu_paillier_random_t) * count,
                        hipMemcpyHostToDevice));

  // kernel_paillier_enc<params> << <(count + IPB - 1) / IPB, TPB >> > (report,
  // gpu_result, gpu_pub, gpu_pt, gpu_random, count);
  unsigned int ps, rep, rem, i = 0;
  ps = TPB * BPG;  // kernel parallel
  if (ps < count) {
    rep = count / ps;
    for (i = 0; i < rep; i++) {
      kernel_paillier_enc<params><<<(ps + IPB - 1) / IPB, TPB>>>(
          report, &gpu_result[i * ps], gpu_pub, &gpu_pt[i * ps],
          &gpu_random[i * ps], ps);
    }
    rem = count - ps * rep;
    kernel_paillier_enc<params><<<(rem + IPB - 1) / IPB, TPB>>>(
        report, &gpu_result[i * ps], gpu_pub, &gpu_pt[i * ps],
        &gpu_random[i * ps], rem);
  } else {
    kernel_paillier_enc<params><<<(count + IPB - 1) / IPB, TPB>>>(
        report, gpu_result, gpu_pub, gpu_pt, gpu_random, count);
  }

  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_LAST_CHECK();
  CGBN_CHECK(report);
  CUDA_CHECK(hipMemcpy(res, gpu_result,
                        sizeof(gpu_paillier_ciphertext_t) * count,
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(gpu_result));
  CUDA_CHECK(hipFree(gpu_pub));
  CUDA_CHECK(hipFree(gpu_pt));
  CUDA_CHECK(hipFree(gpu_random));
  CUDA_CHECK(cgbn_error_report_free(report));
  return 0;
}

int gpu_paillier_dec(h_paillier_plaintext_t *res, h_paillier_pubkey_t *pub,
                     h_paillier_prvkey_t *prv, h_paillier_ciphertext_t *ct,
                     unsigned int count) {
  // unsigned int TPI, TPB, IPB;
  int32_t TPB = (params::TPB == 0)
                    ? 64
                    : params::TPB;  // default threads per block to 128
  int32_t TPI = params::TPI, IPB = TPB / TPI;
  gpu_paillier_plaintext_t *gpu_result;
  gpu_paillier_pubkey_t *gpu_pub;
  gpu_paillier_prvkey_t *gpu_prv;
  gpu_paillier_ciphertext_t *gpu_ct;

  cgbn_error_report_t *report;

  CUDA_CHECK(hipMalloc((void **)&gpu_result,
                        sizeof(gpu_paillier_plaintext_t) * count));
  CUDA_CHECK(hipMalloc((void **)&gpu_pub, sizeof(gpu_paillier_pubkey_t)));
  CUDA_CHECK(hipMalloc((void **)&gpu_prv, sizeof(gpu_paillier_prvkey_t)));
  CUDA_CHECK(
      hipMalloc((void **)&gpu_ct, sizeof(gpu_paillier_ciphertext_t) * count));

  CUDA_CHECK(cgbn_error_report_alloc(&report));
  CUDA_CHECK(hipMemcpy(gpu_pub, pub, sizeof(gpu_paillier_pubkey_t),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(gpu_prv, prv, sizeof(gpu_paillier_prvkey_t),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(gpu_ct, ct, sizeof(gpu_paillier_ciphertext_t) * count,
                        hipMemcpyHostToDevice));

  kernel_paillier_dec<params><<<(count + IPB - 1) / IPB, TPB>>>(
      report, gpu_result, gpu_pub, gpu_prv, gpu_ct, count);

  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_LAST_CHECK();
  CGBN_CHECK(report);
  CUDA_CHECK(hipMemcpy(res, gpu_result,
                        sizeof(gpu_paillier_plaintext_t) * count,
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(gpu_result));
  CUDA_CHECK(hipFree(gpu_pub));
  CUDA_CHECK(hipFree(gpu_prv));
  CUDA_CHECK(hipFree(gpu_ct));
  CUDA_CHECK(cgbn_error_report_free(report));
  return 0;
}

int gpu_paillier_e_add(h_paillier_pubkey_t *pub, h_paillier_ciphertext_t *res,
                       h_paillier_ciphertext_t *ct0,
                       h_paillier_ciphertext_t *ct1, unsigned int count) {
  // unsigned int TPI, TPB, IPB;
  int32_t TPB = (params::TPB == 0)
                    ? 64
                    : params::TPB;  // default threads per block to 128
  int32_t TPI = params::TPI, IPB = TPB / TPI;
  gpu_paillier_ciphertext_t *gpu_result;
  gpu_paillier_pubkey_t *gpu_pub;
  gpu_paillier_ciphertext_t *gpu_ct0;
  gpu_paillier_ciphertext_t *gpu_ct1;

  cgbn_error_report_t *report;

  CUDA_CHECK(hipMalloc((void **)&gpu_result,
                        sizeof(gpu_paillier_ciphertext_t) * count));
  CUDA_CHECK(hipMalloc((void **)&gpu_pub, sizeof(gpu_paillier_pubkey_t) * 1));
  CUDA_CHECK(
      hipMalloc((void **)&gpu_ct0, sizeof(gpu_paillier_ciphertext_t) * count));
  CUDA_CHECK(
      hipMalloc((void **)&gpu_ct1, sizeof(gpu_paillier_ciphertext_t) * count));

  CUDA_CHECK(cgbn_error_report_alloc(&report));
  CUDA_CHECK(hipMemcpy(gpu_pub, pub, sizeof(gpu_paillier_pubkey_t) * 1,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(gpu_ct0, ct0, sizeof(gpu_paillier_ciphertext_t) * count,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(gpu_ct1, ct1, sizeof(gpu_paillier_ciphertext_t) * count,
                        hipMemcpyHostToDevice));

  kernel_paillier_e_add<params><<<(count + IPB - 1) / IPB, TPB>>>(
      report, gpu_result, gpu_pub, gpu_ct0, gpu_ct1, count);

  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_LAST_CHECK();
  CGBN_CHECK(report);
  CUDA_CHECK(hipMemcpy(res, gpu_result,
                        sizeof(gpu_paillier_ciphertext_t) * count,
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(gpu_result));
  CUDA_CHECK(hipFree(gpu_pub));
  CUDA_CHECK(hipFree(gpu_ct0));
  CUDA_CHECK(hipFree(gpu_ct1));
  CUDA_CHECK(cgbn_error_report_free(report));
  return 0;
}

int gpu_paillier_e_inverse(h_paillier_pubkey_t *pub,
                           h_paillier_ciphertext_t *res,
                           h_paillier_ciphertext_t *ct, unsigned int count) {
  // unsigned int TPI, TPB, IPB;
  int32_t TPB = (params::TPB == 0)
                    ? 64
                    : params::TPB;  // default threads per block to 128
  int32_t TPI = params::TPI, IPB = TPB / TPI;
  gpu_paillier_ciphertext_t *gpu_result;
  gpu_paillier_pubkey_t *gpu_pub;
  gpu_paillier_ciphertext_t *gpu_ct;

  cgbn_error_report_t *report;

  CUDA_CHECK(hipMalloc((void **)&gpu_result,
                        sizeof(gpu_paillier_ciphertext_t) * count));
  CUDA_CHECK(hipMalloc((void **)&gpu_pub, sizeof(gpu_paillier_pubkey_t) * 1));
  CUDA_CHECK(
      hipMalloc((void **)&gpu_ct, sizeof(gpu_paillier_ciphertext_t) * count));

  CUDA_CHECK(cgbn_error_report_alloc(&report));
  CUDA_CHECK(hipMemcpy(gpu_pub, pub, sizeof(gpu_paillier_pubkey_t) * 1,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(gpu_ct, ct, sizeof(gpu_paillier_ciphertext_t) * count,
                        hipMemcpyHostToDevice));

  kernel_paillier_inv<params><<<(count + IPB - 1) / IPB, TPB>>>(
      report, gpu_result, gpu_pub, gpu_ct, count);

  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_LAST_CHECK();
  CGBN_CHECK(report);
  CUDA_CHECK(hipMemcpy(res, gpu_result,
                        sizeof(gpu_paillier_ciphertext_t) * count,
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(gpu_result));
  CUDA_CHECK(hipFree(gpu_pub));
  CUDA_CHECK(hipFree(gpu_ct));
  CUDA_CHECK(cgbn_error_report_free(report));
  return 0;
}

int gpu_paillier_e_add_const(h_paillier_pubkey_t *pub,
                             h_paillier_ciphertext_t *res,
                             h_paillier_ciphertext_t *ct,
                             h_paillier_plaintext_t *constant,
                             unsigned int count) {
  int32_t TPB = (params::TPB == 0)
                    ? 64
                    : params::TPB;  // default threads per block to 128
  int32_t TPI = params::TPI, IPB = TPB / TPI;

  gpu_paillier_pubkey_t *gpu_pub;
  gpu_paillier_ciphertext_t *gpu_result;
  gpu_paillier_ciphertext_t *gpu_ct;
  gpu_paillier_plaintext_t *gpu_constant;

  cgbn_error_report_t *report;

  gpu_constant = (gpu_paillier_plaintext_t *)constant;

  CUDA_CHECK(hipMalloc((void **)&gpu_result,
                        sizeof(gpu_paillier_ciphertext_t) * count));
  CUDA_CHECK(hipMalloc((void **)&gpu_pub, sizeof(gpu_paillier_pubkey_t)));
  CUDA_CHECK(
      hipMalloc((void **)&gpu_ct, sizeof(gpu_paillier_ciphertext_t) * count));
  CUDA_CHECK(hipMalloc((void **)&gpu_constant,
                        sizeof(gpu_paillier_plaintext_t) * count));

  CUDA_CHECK(cgbn_error_report_alloc(&report));

  CUDA_CHECK(hipMemcpy(gpu_pub, pub, sizeof(gpu_paillier_pubkey_t),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(gpu_ct, ct, sizeof(gpu_paillier_ciphertext_t) * count,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(gpu_constant, constant,
                        sizeof(gpu_paillier_ciphertext_t) * count,
                        hipMemcpyHostToDevice));

  kernel_paillier_e_add_const<params><<<(count + IPB - 1) / IPB, TPB>>>(
      report, gpu_result, gpu_pub, gpu_ct, gpu_constant, count);

  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_LAST_CHECK();
  CGBN_CHECK(report);
  CUDA_CHECK(hipMemcpy(res, gpu_result,
                        sizeof(gpu_paillier_ciphertext_t) * count,
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(gpu_result));
  CUDA_CHECK(hipFree(gpu_pub));
  CUDA_CHECK(hipFree(gpu_ct));
  CUDA_CHECK(hipFree(gpu_constant));
  CUDA_CHECK(cgbn_error_report_free(report));
  return 0;
}

int gpu_paillier_sub_ct(h_paillier_pubkey_t *pub, h_paillier_ciphertext_t *res,
                        h_paillier_ciphertext_t *ct0,
                        h_paillier_ciphertext_t *ct1, unsigned int count) {
  // unsigned int TPI, TPB, IPB;
  int32_t TPB = (params::TPB == 0)
                    ? 64
                    : params::TPB;  // default threads per block to 128
  int32_t TPI = params::TPI, IPB = TPB / TPI;
  gpu_paillier_ciphertext_t *gpu_result;
  gpu_paillier_pubkey_t *gpu_pub;
  gpu_paillier_ciphertext_t *gpu_ct0;
  gpu_paillier_ciphertext_t *gpu_ct1;

  cgbn_error_report_t *report;

  CUDA_CHECK(hipMalloc((void **)&gpu_result,
                        sizeof(gpu_paillier_ciphertext_t) * count));
  CUDA_CHECK(hipMalloc((void **)&gpu_pub, sizeof(gpu_paillier_pubkey_t) * 1));
  CUDA_CHECK(
      hipMalloc((void **)&gpu_ct0, sizeof(gpu_paillier_ciphertext_t) * count));
  CUDA_CHECK(
      hipMalloc((void **)&gpu_ct1, sizeof(gpu_paillier_ciphertext_t) * count));

  CUDA_CHECK(cgbn_error_report_alloc(&report));
  CUDA_CHECK(hipMemcpy(gpu_pub, pub, sizeof(gpu_paillier_pubkey_t) * 1,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(gpu_ct0, ct0, sizeof(gpu_paillier_ciphertext_t) * count,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(gpu_ct1, ct1, sizeof(gpu_paillier_ciphertext_t) * count,
                        hipMemcpyHostToDevice));

  kernel_paillier_e_sub<params><<<(count + IPB - 1) / IPB, TPB>>>(
      report, gpu_result, gpu_pub, gpu_ct0, gpu_ct1, count);

  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_LAST_CHECK();
  CGBN_CHECK(report);
  CUDA_CHECK(hipMemcpy(res, gpu_result,
                        sizeof(gpu_paillier_ciphertext_t) * count,
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(gpu_result));
  CUDA_CHECK(hipFree(gpu_pub));
  CUDA_CHECK(hipFree(gpu_ct0));
  CUDA_CHECK(hipFree(gpu_ct1));
  CUDA_CHECK(cgbn_error_report_free(report));
  return 0;
}

int gpu_paillier_sub_ctpt(h_paillier_pubkey_t *pub,
                          h_paillier_ciphertext_t *res,
                          h_paillier_ciphertext_t *ct,
                          h_paillier_plaintext_t *pt, unsigned int count) {
  // unsigned int TPI, TPB, IPB;
  int32_t TPB = (params::TPB == 0)
                    ? 64
                    : params::TPB;  // default threads per block to 128
  int32_t TPI = params::TPI, IPB = TPB / TPI;
  gpu_paillier_ciphertext_t *gpu_result;
  gpu_paillier_pubkey_t *gpu_pub;
  gpu_paillier_ciphertext_t *gpu_ct;
  gpu_paillier_plaintext_t *gpu_pt;

  cgbn_error_report_t *report;

  CUDA_CHECK(hipMalloc((void **)&gpu_result,
                        sizeof(gpu_paillier_ciphertext_t) * count));
  CUDA_CHECK(hipMalloc((void **)&gpu_pub, sizeof(gpu_paillier_pubkey_t) * 1));
  CUDA_CHECK(
      hipMalloc((void **)&gpu_ct, sizeof(gpu_paillier_ciphertext_t) * count));
  CUDA_CHECK(
      hipMalloc((void **)&gpu_pt, sizeof(gpu_paillier_plaintext_t) * count));

  CUDA_CHECK(cgbn_error_report_alloc(&report));
  CUDA_CHECK(hipMemcpy(gpu_pub, pub, sizeof(gpu_paillier_pubkey_t) * 1,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(gpu_ct, ct, sizeof(gpu_paillier_ciphertext_t) * count,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(gpu_pt, pt, sizeof(gpu_paillier_plaintext_t) * count,
                        hipMemcpyHostToDevice));

  kernel_paillier_e_sub_ctpt<params><<<(count + IPB - 1) / IPB, TPB>>>(
      report, gpu_result, gpu_pub, gpu_ct, gpu_pt, count);

  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_LAST_CHECK();
  CGBN_CHECK(report);
  CUDA_CHECK(hipMemcpy(res, gpu_result,
                        sizeof(gpu_paillier_ciphertext_t) * count,
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(gpu_result));
  CUDA_CHECK(hipFree(gpu_pub));
  CUDA_CHECK(hipFree(gpu_ct));
  CUDA_CHECK(hipFree(gpu_pt));
  CUDA_CHECK(cgbn_error_report_free(report));
  return 0;
}

int gpu_paillier_sub_ptct(h_paillier_pubkey_t *pub,
                          h_paillier_ciphertext_t *res,
                          h_paillier_plaintext_t *pt,
                          h_paillier_ciphertext_t *ct, unsigned int count) {
  // unsigned int TPI, TPB, IPB;
  int32_t TPB = (params::TPB == 0)
                    ? 64
                    : params::TPB;  // default threads per block to 128
  int32_t TPI = params::TPI, IPB = TPB / TPI;
  gpu_paillier_ciphertext_t *gpu_result;
  gpu_paillier_pubkey_t *gpu_pub;
  gpu_paillier_ciphertext_t *gpu_ct;
  gpu_paillier_plaintext_t *gpu_pt;

  cgbn_error_report_t *report;

  CUDA_CHECK(hipMalloc((void **)&gpu_result,
                        sizeof(gpu_paillier_ciphertext_t) * count));
  CUDA_CHECK(hipMalloc((void **)&gpu_pub, sizeof(gpu_paillier_pubkey_t) * 1));
  CUDA_CHECK(
      hipMalloc((void **)&gpu_ct, sizeof(gpu_paillier_ciphertext_t) * count));
  CUDA_CHECK(
      hipMalloc((void **)&gpu_pt, sizeof(gpu_paillier_plaintext_t) * count));

  CUDA_CHECK(cgbn_error_report_alloc(&report));
  CUDA_CHECK(hipMemcpy(gpu_pub, pub, sizeof(gpu_paillier_pubkey_t) * 1,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(gpu_ct, ct, sizeof(gpu_paillier_ciphertext_t) * count,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(gpu_pt, pt, sizeof(gpu_paillier_plaintext_t) * count,
                        hipMemcpyHostToDevice));

  kernel_paillier_e_sub_ptct<params><<<(count + IPB - 1) / IPB, TPB>>>(
      report, gpu_result, gpu_pub, gpu_pt, gpu_ct, count);

  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_LAST_CHECK();
  CGBN_CHECK(report);
  CUDA_CHECK(hipMemcpy(res, gpu_result,
                        sizeof(gpu_paillier_ciphertext_t) * count,
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(gpu_result));
  CUDA_CHECK(hipFree(gpu_pub));
  CUDA_CHECK(hipFree(gpu_ct));
  CUDA_CHECK(hipFree(gpu_pt));
  CUDA_CHECK(cgbn_error_report_free(report));
  return 0;
}

int gpu_paillier_e_mul_const(h_paillier_pubkey_t *pub,
                             h_paillier_ciphertext_t *res,
                             h_paillier_ciphertext_t *ct,
                             h_paillier_plaintext_t *constant,
                             unsigned int count) {
  int32_t TPB = (params::TPB == 0)
                    ? 64
                    : params::TPB;  // default threads per block to 128
  int32_t TPI = params::TPI, IPB = TPB / TPI;
  gpu_paillier_ciphertext_t *gpu_result;
  gpu_paillier_pubkey_t *gpu_pub;
  gpu_paillier_ciphertext_t *gpu_ct;
  gpu_paillier_plaintext_t *gpu_constant;

  cgbn_error_report_t *report;

  gpu_constant = (gpu_paillier_plaintext_t *)constant;
  CUDA_CHECK(hipMalloc((void **)&gpu_result,
                        sizeof(gpu_paillier_ciphertext_t) * count));
  CUDA_CHECK(hipMalloc((void **)&gpu_pub, sizeof(gpu_paillier_pubkey_t)));
  CUDA_CHECK(
      hipMalloc((void **)&gpu_ct, sizeof(gpu_paillier_ciphertext_t) * count));
  CUDA_CHECK(hipMalloc((void **)&gpu_constant,
                        sizeof(gpu_paillier_plaintext_t) * count));

  CUDA_CHECK(cgbn_error_report_alloc(&report));
  CUDA_CHECK(hipMemcpy(gpu_pub, (gpu_paillier_pubkey_t *)pub,
                        sizeof(gpu_paillier_pubkey_t), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(gpu_ct, (gpu_paillier_ciphertext_t *)ct,
                        sizeof(gpu_paillier_ciphertext_t) * count,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(gpu_constant, constant,
                        sizeof(gpu_paillier_plaintext_t) * count,
                        hipMemcpyHostToDevice));

  kernel_paillier_e_mul_const<params><<<(count + IPB - 1) / IPB, TPB>>>(
      report, gpu_result, gpu_pub, gpu_ct, gpu_constant, count);

  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_LAST_CHECK();
  CGBN_CHECK(report);
  CUDA_CHECK(hipMemcpy(res, gpu_result,
                        sizeof(gpu_paillier_ciphertext_t) * count,
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(gpu_result));
  CUDA_CHECK(hipFree(gpu_pub));
  CUDA_CHECK(hipFree(gpu_ct));
  CUDA_CHECK(hipFree(gpu_constant));
  CUDA_CHECK(cgbn_error_report_free(report));
  return 0;
}

int gpu_paillier_compare(h_paillier_plaintext_t *plain, unsigned int *res,
                         unsigned int count) {
  int32_t TPB = (params::TPB == 0)
                    ? 64
                    : params::TPB;  // default threads per block to 128
  int32_t TPI = params::TPI, IPB = TPB / TPI;  // IPB is instances per block

  gpu_paillier_plaintext_t *gpu_plain;
  unsigned int *gpu_res;

  cgbn_error_report_t *report;

  CUDA_CHECK(hipMalloc((void **)&gpu_plain,
                        sizeof(gpu_paillier_plaintext_t) * count));
  CUDA_CHECK(hipMalloc((void **)&gpu_res, sizeof(unsigned int) * count));

  CUDA_CHECK(cgbn_error_report_alloc(&report));
  CUDA_CHECK(hipMemcpy(gpu_plain, plain,
                        sizeof(gpu_paillier_plaintext_t) * count,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(gpu_res, res, sizeof(unsigned int) * count,
                        hipMemcpyHostToDevice));

  kernel_paillier_compare<params>
      <<<(count + IPB - 1) / IPB, TPB>>>(report, gpu_plain, gpu_res, count);

  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_LAST_CHECK();
  CGBN_CHECK(report);
  CUDA_CHECK(hipFree(gpu_plain));
  CUDA_CHECK(hipFree(gpu_res));
  CUDA_CHECK(cgbn_error_report_free(report));
  return 0;
}
