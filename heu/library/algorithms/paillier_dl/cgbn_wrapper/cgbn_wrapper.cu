#include "hip/hip_runtime.h"
#include <gmp.h>
#include "cgbn/cgbn.h"
// #include "cgbn_wrapper_defs.h"
#include "cgbn_wrapper.h"
#include "gpu_support.h"

#define DEBUG

namespace heu::lib::algorithms::paillier_z {

typedef cgbn_context_t<TPI>         context_t;
typedef cgbn_env_t<context_t, BITS> env_t;
typedef typename env_t::cgbn_t                bn_t;
typedef typename env_t::cgbn_local_t          bn_local_t;
typedef cgbn_mem_t<BITS> gpu_mpz; 

void p_mpint(char *name, MPInt *d) {
  printf("[%s]\n", name);
  for (int i=0; i<(d->SizeUsed() + 3) / 4; i++) {
    printf("%08x ", ((uint32_t *)(d->n_.dp))[i]);
  }
  printf("\n");
}

__device__ void p_cgbn(char *name, cgbn_mem_t<BITS> *d) {
  printf("[%s]\n", name);
  for (int i=0; i<(sizeof(d->_limbs) + 3) / 4; i++) {
    printf("%08x ", d->_limbs[i]);
  }
  printf("\n");
}

static void p_mpz(char *name, mpz_t d) {
  printf("[%s]\n", name);
  for (int i=0; i<d->_mp_alloc * 2; i++) {
    printf("%08x ", ((uint32_t *)(d->_mp_d))[i]);
  }
  printf("\n");
}

void mpint_cal_used(MPInt* out) {
  int used = 0;
  for (int i=0; i<out->SizeAllocated(); i++) {
    if (((unsigned char *)out->n_.dp)[i] != 0) {
      used = i + 1;
    }
  }
  out->n_.used = (used + sizeof(mp_digit) - 1) / sizeof(mp_digit);
}

void mpint_handle_neg(MPInt* out) {
  int used = out->SizeUsed();
  if ((((unsigned char *)out->n_.dp)[used - 1] & 0x80) > 0 || out->n_.sign == MP_NEG) {
    for (int i=0; i<used; i++) {
      ((unsigned char *)out->n_.dp)[i] = ~((unsigned char *)out->n_.dp)[i];
    }
    out->n_.sign = MP_NEG;
  }
}

void pt_cal_used(Plaintext* out) {
  int used = 0;
  for (int i=0; i<out->SizeAllocated(); i++) {
    if (((unsigned char *)out->n_.dp)[i] != 0) {
      used = i + 1;
    }
  }
  out->n_.used = (used + sizeof(mp_digit) - 1) / sizeof(mp_digit);
}

void pt_handle_neg(Plaintext* out) {
  int used = out->SizeUsed();
  if ((((unsigned char *)out->n_.dp)[used - 1] & 0x80) > 0 || out->n_.sign == MP_NEG) {
    for (int i=0; i<used; i++) {
      ((unsigned char *)out->n_.dp)[i] = ~((unsigned char *)out->n_.dp)[i];
    }
    out->n_.sign = MP_NEG;
  }
}

void store2gmp(mpz_t z, cgbn_mem_t<BITS> *address ) {
  mpz_import(z, (BITS+31)/32, -1, sizeof(uint32_t), 0, 0, (uint32_t *)address);
}

__device__  __forceinline__ void l_func(env_t &bn_env, env_t::cgbn_t &out, env_t::cgbn_t &cipher_t, env_t::cgbn_t &x_t, env_t::cgbn_t &xsquare_t, env_t::cgbn_t &hx_t) {
  env_t::cgbn_t  tmp, tmp2, cipher_lt;
  cgbn_sub_ui32(bn_env, tmp2, x_t, 1);
  if(cgbn_compare(bn_env, cipher_t, xsquare_t) >= 0) {
    cgbn_rem(bn_env, cipher_lt, cipher_t, xsquare_t);
    cgbn_modular_power(bn_env, tmp, cipher_lt, tmp2, xsquare_t);
  } else {
    cgbn_modular_power(bn_env, tmp, cipher_t, tmp2, xsquare_t);
  }
  cgbn_sub_ui32(bn_env, tmp, tmp, 1);
  cgbn_div(bn_env, tmp, tmp, x_t);
  cgbn_mul(bn_env, tmp, tmp, hx_t);
  cgbn_rem(bn_env, tmp, tmp, x_t);
  cgbn_set(bn_env, out, tmp);
}

__global__ __noinline__ void raw_encrypt(PublicKey *pub_key, cgbn_error_report_t *report, gpu_mpz *plains, gpu_mpz *ciphers,int count, int rand_seed ) {
  int tid=(blockIdx.x*blockDim.x + threadIdx.x)/TPI;
  if(tid>=count)
    return;
  context_t      bn_context(cgbn_report_monitor, report, tid);  
  env_t          bn_env(bn_context.env<env_t>());                   
  env_t::cgbn_t  n, nsquare, plain,  tmp, max_int, neg_plain, neg_cipher, cipher, r;               
  cgbn_load(bn_env, n, (cgbn_mem_t<BITS> *)pub_key->dev_n_);      
  cgbn_load(bn_env, plain, plains + tid);      
  cgbn_load(bn_env, nsquare, (cgbn_mem_t<BITS> *)pub_key->dev_nsquare_);
  cgbn_load(bn_env, max_int, (cgbn_mem_t<BITS> *)pub_key->dev_max_int_);
  cgbn_load(bn_env, plain, plains + tid);
  cgbn_sub(bn_env, tmp, n, max_int); 
  if(cgbn_compare(bn_env, plain, tmp) >= 0 &&  cgbn_compare(bn_env, plain, n) < 0) {
    // Very large plaintext, take a sneaky shortcut using inverses
    cgbn_sub(bn_env, neg_plain, n, plain);
    cgbn_mul(bn_env, neg_cipher, n, neg_plain);
    cgbn_add_ui32(bn_env, neg_cipher, neg_cipher, 1);
    cgbn_rem(bn_env, neg_cipher, neg_cipher, nsquare);
    cgbn_modular_inverse(bn_env, cipher, neg_cipher, nsquare);
  } else {
    cgbn_mul(bn_env, cipher, n, plain);
    cgbn_add_ui32(bn_env, cipher, cipher, 1);
    cgbn_rem(bn_env, cipher, cipher, nsquare);
  }
  cgbn_set_ui32(bn_env, r, rand_seed);
  cgbn_modular_power(bn_env, tmp, r, n, nsquare); 
  cgbn_mul(bn_env, tmp, cipher, tmp); 
  cgbn_rem(bn_env, r, tmp, nsquare);
  cgbn_store(bn_env, ciphers + tid, r);   // store r into sum

#ifdef DEBUG
  if (blockIdx.x == 0 && threadIdx.x == 0) {
    p_cgbn("[encrypt] dev_plains", plains);
    p_cgbn("[encrypt] dev_ciphers", ciphers);
  }
#endif
}

void CGBNWrapper::Encrypt(const MPInt m, const PublicKey pk,  MPInt &rn, Ciphertext &ct) {
  int32_t              TPB=128;
  int32_t              IPB=TPB/TPI;
  int count = 1;

  mpint_handle_neg(const_cast<MPInt *>(&m));

  cgbn_error_report_t *report;
  cgbn_mem_t<BITS> *dev_plains;
  cgbn_mem_t<BITS> *dev_ciphers;
  PublicKey *dev_pub_key;

  CUDA_CHECK(hipMalloc((void **)&dev_plains, sizeof(cgbn_mem_t<BITS>)));
  CUDA_CHECK(hipMalloc((void **)&dev_ciphers, sizeof(cgbn_mem_t<BITS>)));
  CUDA_CHECK(hipMalloc((void **)&dev_pub_key, sizeof(pk))); 
  CUDA_CHECK(hipMemcpy(dev_plains->_limbs, m.n_.dp, const_cast<MPInt *>(&m)->SizeUsed(), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(dev_pub_key, &pk,  sizeof(pk), hipMemcpyHostToDevice));
  CUDA_CHECK(cgbn_error_report_alloc(&report));

  raw_encrypt<<<(count+IPB-1)/IPB, TPB>>>(dev_pub_key, report,  dev_plains, dev_ciphers, count, 12345); 
  CUDA_CHECK(hipDeviceSynchronize());
  if (ct.c_.SizeAllocated() > sizeof(dev_ciphers->_limbs)) {
    printf("%s:%d No enough memory, need: %d, real: %d\n", __FILE__, __LINE__, sizeof(dev_ciphers->_limbs), ct.c_.SizeAllocated());
    abort();
  }
  CUDA_CHECK(hipMemcpy(ct.c_.n_.dp, dev_ciphers->_limbs, ct.c_.SizeAllocated(), hipMemcpyDeviceToHost)); 

  CGBN_CHECK(report);

  CUDA_CHECK(cgbn_error_report_free(report));
  CUDA_CHECK(hipFree(dev_plains));
  CUDA_CHECK(hipFree(dev_ciphers));
  CUDA_CHECK(hipFree(dev_pub_key));
}

void CGBNWrapper::Encrypt(absl::Span<const Plaintext> pts, const PublicKey pk, std::vector<MPInt> &rns, std::vector<Ciphertext> &cts) {
  int32_t              TPB=128;
  int32_t              IPB=TPB/TPI;
  int count = pts.size();

  std::vector<Plaintext> handled_pts;
  for (int i=0; i<count; i++) {
    Plaintext pt = pts[i];
    pt_handle_neg(&pt);
    handled_pts.push_back(pt);
  }

  cgbn_error_report_t *report;
  cgbn_mem_t<BITS> *dev_plains;
  cgbn_mem_t<BITS> *dev_ciphers;
  PublicKey *dev_pub_key;

  CUDA_CHECK(hipMalloc((void **)&dev_plains, sizeof(cgbn_mem_t<BITS>) * count));
  CUDA_CHECK(hipMalloc((void **)&dev_ciphers, sizeof(cgbn_mem_t<BITS>) * count));
  CUDA_CHECK(hipMalloc((void **)&dev_pub_key, sizeof(pk))); 
  for (int i=0; i<count; i++) {
    CUDA_CHECK(hipMemcpy(dev_plains[i]._limbs, handled_pts[i].n_.dp, handled_pts[i].SizeUsed(), hipMemcpyHostToDevice));
  }
  CUDA_CHECK(hipMemcpy(dev_pub_key, &pk,  sizeof(pk), hipMemcpyHostToDevice));
  CUDA_CHECK(cgbn_error_report_alloc(&report));

  raw_encrypt<<<(count+IPB-1)/IPB, TPB>>>(dev_pub_key, report,  dev_plains, dev_ciphers, count, 12345); 
  CUDA_CHECK(hipDeviceSynchronize());

  for (int i=0; i<count; i++) {
    if (cts[i].c_.SizeAllocated() > sizeof(dev_ciphers[i]._limbs)) {
      printf("%s:%d No enough memory, need: %d, real: %d\n", __FILE__, __LINE__, sizeof(dev_ciphers[i]._limbs), cts[i].c_.SizeAllocated());
      abort();
    }
    CUDA_CHECK(hipMemcpy(cts[i].c_.n_.dp, dev_ciphers[i]._limbs, cts[i].c_.SizeAllocated(), hipMemcpyDeviceToHost)); 
  }

  CGBN_CHECK(report);

  CUDA_CHECK(cgbn_error_report_free(report));
  CUDA_CHECK(hipFree(dev_plains));
  CUDA_CHECK(hipFree(dev_ciphers));
  CUDA_CHECK(hipFree(dev_pub_key));
}


__global__ void raw_decrypt(SecretKey *priv_key, dev_mem_t<BITS> *pk_n, cgbn_error_report_t *report, gpu_mpz *plains, gpu_mpz *ciphers, int count) {
  int tid=(blockIdx.x*blockDim.x + threadIdx.x)/TPI;
  if(tid>=count)
    return;

  context_t      bn_context(cgbn_report_monitor, report, tid);
  env_t          bn_env(bn_context.env<env_t>());
  env_t::cgbn_t  mp, mq, tmp, q_inverse, n, p, q, hp, hq, psquare, qsquare, cipher;
  cgbn_load(bn_env, cipher, ciphers + tid);
  cgbn_load(bn_env, q_inverse, (cgbn_mem_t<BITS> *)priv_key->dev_q_inverse_);
  cgbn_load(bn_env, n, (cgbn_mem_t<BITS> *)pk_n);
  cgbn_load(bn_env, p, (cgbn_mem_t<BITS> *)priv_key->dev_p_);
  cgbn_load(bn_env, q, (cgbn_mem_t<BITS> *)priv_key->dev_q_);
  cgbn_load(bn_env, hp, (cgbn_mem_t<BITS> *)priv_key->dev_hp_);
  cgbn_load(bn_env, hq, (cgbn_mem_t<BITS> *)priv_key->dev_hq_);
  cgbn_load(bn_env, psquare, (cgbn_mem_t<BITS> *)priv_key->dev_psquare_);
  cgbn_load(bn_env, qsquare, (cgbn_mem_t<BITS> *)priv_key->dev_qsquare_);
  l_func(bn_env, mp, cipher, p, psquare, hp); 
  l_func(bn_env, mq, cipher, q, qsquare, hq); 
  cgbn_sub(bn_env, tmp, mp, mq);
  cgbn_mul(bn_env, tmp, tmp, q_inverse); 
  cgbn_rem(bn_env, tmp, tmp, p);
  cgbn_mul(bn_env, tmp, tmp, q);
  cgbn_add(bn_env, tmp, mq, tmp);
  cgbn_rem(bn_env, tmp, tmp, n);
  cgbn_store(bn_env, plains + tid, tmp);

#ifdef DEBUG
  if (blockIdx.x == 0 && threadIdx.x == 0) {
    p_cgbn("[decrypt] dev_plains", plains);
    p_cgbn("[decrypt] dev_ciphers", ciphers);
    p_cgbn("dev_pk_n", (cgbn_mem_t<BITS> *)pk_n);
    p_cgbn("dev_p_", (cgbn_mem_t<BITS> *)priv_key->dev_p_);
    p_cgbn("dev_q_", (cgbn_mem_t<BITS> *)priv_key->dev_q_);
    p_cgbn("dev_hp_", (cgbn_mem_t<BITS> *)priv_key->dev_hp_);
    p_cgbn("dev_hq_", (cgbn_mem_t<BITS> *)priv_key->dev_hq_);
    p_cgbn("dev_psquare_", (cgbn_mem_t<BITS> *)priv_key->dev_psquare_);
    p_cgbn("dev_qsquare_", (cgbn_mem_t<BITS> *)priv_key->dev_qsquare_);
  }
#endif
} 

void CGBNWrapper::Decrypt(const Ciphertext& ct, const SecretKey sk, const PublicKey pk, MPInt* out) {
  int32_t              TPB=128;
  int32_t              IPB=TPB/TPI;
  int count = 1;

  cgbn_error_report_t *report;
  cgbn_mem_t<BITS> *dev_plains;
  cgbn_mem_t<BITS> *dev_ciphers;
  SecretKey *dev_priv_key;
  cgbn_mem_t<BITS> cpu_ciphers;

  CUDA_CHECK(hipMalloc((void **)&dev_plains, sizeof(cgbn_mem_t<BITS>)));
  CUDA_CHECK(hipMalloc((void **)&dev_ciphers, sizeof(cgbn_mem_t<BITS>)));
  CUDA_CHECK(hipMalloc((void **)&dev_priv_key, sizeof(sk))); 
  CUDA_CHECK(hipMemcpy(dev_ciphers->_limbs, ct.c_.n_.dp, const_cast<MPInt *>(&ct.c_)->SizeUsed(), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(dev_priv_key, &sk,  sizeof(sk), hipMemcpyHostToDevice));
  CUDA_CHECK(cgbn_error_report_alloc(&report));

  raw_decrypt<<<(count+IPB-1)/IPB, TPB>>>(dev_priv_key,  const_cast<PublicKey *>(&pk)->dev_n_, report, dev_plains, dev_ciphers, count);
  CUDA_CHECK(hipDeviceSynchronize());
  if (out->SizeAllocated() > sizeof(dev_plains->_limbs)) {
    printf("%s:%d No enough memory, need: %d, real: %d\n", __FILE__, __LINE__, sizeof(dev_plains->_limbs), out->SizeAllocated());
    abort();
  }
  
  CUDA_CHECK(hipMemcpy(out->n_.dp, dev_plains->_limbs, out->SizeAllocated(), hipMemcpyDeviceToHost)); 
  CGBN_CHECK(report);

  mpint_cal_used(out);
  mpint_handle_neg(out);

  CUDA_CHECK(cgbn_error_report_free(report));
  CUDA_CHECK(hipFree(dev_plains));
  CUDA_CHECK(hipFree(dev_ciphers));
  CUDA_CHECK(hipFree(dev_priv_key));
}

void CGBNWrapper::Decrypt(absl::Span<const Ciphertext>& cts, const SecretKey sk, const PublicKey pk, absl::Span<Plaintext>* pts) {
  int32_t              TPB=128;
  int32_t              IPB=TPB/TPI;
  int count = cts.size();

  cgbn_error_report_t *report;
  cgbn_mem_t<BITS> *dev_plains;
  cgbn_mem_t<BITS> *dev_ciphers;
  SecretKey *dev_priv_key;
  cgbn_mem_t<BITS> cpu_ciphers;

  CUDA_CHECK(hipMalloc((void **)&dev_plains, sizeof(cgbn_mem_t<BITS>) * count));
  CUDA_CHECK(hipMalloc((void **)&dev_ciphers, sizeof(cgbn_mem_t<BITS>) * count));
  CUDA_CHECK(hipMalloc((void **)&dev_priv_key, sizeof(sk))); 
  for (int i=0; i<count; i++) {
    Ciphertext ct = cts[i];
    if (ct.c_.SizeUsed() > sizeof(dev_ciphers[i]._limbs)) {
      printf("%s:%d No enough memory, need: %d, real: %d\n", __FILE__, __LINE__, sizeof(dev_ciphers[i]._limbs), ct.c_.SizeUsed());
      abort();
    }   
    CUDA_CHECK(hipMemcpy(dev_ciphers[i]._limbs, ct.c_.n_.dp, ct.c_.SizeUsed(), hipMemcpyHostToDevice));
  }
  CUDA_CHECK(hipMemcpy(dev_priv_key, &sk,  sizeof(sk), hipMemcpyHostToDevice));
  CUDA_CHECK(cgbn_error_report_alloc(&report));

  raw_decrypt<<<(count+IPB-1)/IPB, TPB>>>(dev_priv_key,  const_cast<PublicKey *>(&pk)->dev_n_, report, dev_plains, dev_ciphers, count);
  CUDA_CHECK(hipDeviceSynchronize());
  CGBN_CHECK(report);

  for (int i=0; i<count; i++) {
    MPInt *pt_ptr = &(*pts)[i];
    if (sizeof(dev_plains[i]._limbs) > pt_ptr->SizeAllocated()) {
      mp_init_size(&pt_ptr->n_, sizeof(dev_plains[i]._limbs) / sizeof(uint32_t)); 
      if (sizeof(dev_plains[i]._limbs) > pt_ptr->SizeAllocated()) {
        printf("%s:%d No enough memory, need: %d, real: %d\n", __FILE__, __LINE__, sizeof(dev_plains[i]._limbs), pt_ptr->SizeAllocated());
        abort();
      }
    }    
    CUDA_CHECK(hipMemcpy(pt_ptr->n_.dp, dev_plains[i]._limbs, sizeof(dev_plains[i]._limbs), hipMemcpyDeviceToHost)); 

    mpint_cal_used(pt_ptr);
    mpint_handle_neg(pt_ptr);
  }

  CUDA_CHECK(cgbn_error_report_free(report));
  CUDA_CHECK(hipFree(dev_plains));
  CUDA_CHECK(hipFree(dev_ciphers));
  CUDA_CHECK(hipFree(dev_priv_key));
}


static void store2dev(dev_mem_t<BITS> *address, mpz_t z) {
  if (std::abs(z->_mp_size) * sizeof(mp_limb_t) > sizeof(address->_limbs)) {
    printf("%s:%d No enough memory, need: %d, real: %d\n", __FILE__, __LINE__, std::abs(z->_mp_size)  * sizeof(mp_limb_t), sizeof(address->_limbs));
    abort();
  }
  CUDA_CHECK(hipMemset(address->_limbs, 0, sizeof(address->_limbs)));
  CUDA_CHECK(hipMemcpy(address->_limbs, z->_mp_d, std::abs(z->_mp_size) * sizeof(mp_limb_t), hipMemcpyHostToDevice));
}

void CGBNWrapper::DevMalloc(PublicKey *pk) {
  CUDA_CHECK(hipMalloc((void **)&pk->dev_g_, sizeof(cgbn_mem_t<BITS>)));
  CUDA_CHECK(hipMalloc((void **)&pk->dev_n_, sizeof(cgbn_mem_t<BITS>)));
  CUDA_CHECK(hipMalloc((void **)&pk->dev_nsquare_, sizeof(cgbn_mem_t<BITS>)));
  CUDA_CHECK(hipMalloc((void **)&pk->dev_max_int_, sizeof(cgbn_mem_t<BITS>)));
}

void CGBNWrapper::DevFree(PublicKey *pk) {
  CUDA_CHECK(hipFree(pk->dev_g_));
  CUDA_CHECK(hipFree(pk->dev_n_));
  CUDA_CHECK(hipFree(pk->dev_nsquare_));
  CUDA_CHECK(hipFree(pk->dev_max_int_));
}


void CGBNWrapper::DevMalloc(SecretKey *sk) {
  CUDA_CHECK(hipMalloc((void **)&sk->dev_p_, sizeof(cgbn_mem_t<BITS>)));
  CUDA_CHECK(hipMalloc((void **)&sk->dev_q_, sizeof(cgbn_mem_t<BITS>)));
  CUDA_CHECK(hipMalloc((void **)&sk->dev_psquare_, sizeof(cgbn_mem_t<BITS>)));
  CUDA_CHECK(hipMalloc((void **)&sk->dev_qsquare_, sizeof(cgbn_mem_t<BITS>)));
  CUDA_CHECK(hipMalloc((void **)&sk->dev_q_inverse_, sizeof(cgbn_mem_t<BITS>)));
  CUDA_CHECK(hipMalloc((void **)&sk->dev_hp_, sizeof(cgbn_mem_t<BITS>)));
  CUDA_CHECK(hipMalloc((void **)&sk->dev_hq_, sizeof(cgbn_mem_t<BITS>)));
}

void CGBNWrapper::DevFree(SecretKey *sk) {
  CUDA_CHECK(hipFree(sk->dev_p_));
  CUDA_CHECK(hipFree(sk->dev_q_));
  CUDA_CHECK(hipFree(sk->dev_psquare_));
  CUDA_CHECK(hipFree(sk->dev_qsquare_));
  CUDA_CHECK(hipFree(sk->dev_q_inverse_));
  CUDA_CHECK(hipFree(sk->dev_hp_));
  CUDA_CHECK(hipFree(sk->dev_hq_));
}


void CGBNWrapper::StoreToDev(PublicKey *pk) {
  store2dev(pk->dev_g_, pk->g_);
  store2dev(pk->dev_n_, pk->n_);
  store2dev(pk->dev_nsquare_, pk->nsquare_);
  store2dev(pk->dev_max_int_, pk->max_int_);
}

void CGBNWrapper::StoreToDev(SecretKey *sk) {
  store2dev(sk->dev_p_, sk->p_);
  store2dev(sk->dev_q_, sk->q_);
  store2dev(sk->dev_psquare_, sk->psquare_);
  store2dev(sk->dev_qsquare_, sk->qsquare_);
  store2dev(sk->dev_q_inverse_, sk->q_inverse_);
  store2dev(sk->dev_hp_, sk->hp_);
  store2dev(sk->dev_hq_, sk->hq_);
}

} // namespace heu::lib::algorithms::paillier_z